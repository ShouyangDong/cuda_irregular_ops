
#include <hip/hip_runtime.h>
__global__ void depthwise_convolution(float* input, float* filter, float* output) {
    int tid_x = blockIdx.x * blockDim.x + threadIdx.x;
    int tid_y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if(tid_x < 190 && tid_y < 190) {
        int output_idx = tid_y * 190 + tid_x;
        
        for(int c = 0; c < 3; c++) {
            for(int i = 0; i < 3; i++) {
                for(int j = 0; j < 3; j++) {
                    int input_idx = (tid_y + i) * 192 + (tid_x + j);
                    int filter_idx = c * 3 * 3 + i * 3 + j;
                    
                    output[output_idx] += input[input_idx] * filter[filter_idx];
                }
            }
        }
    }
}


extern "C" void depthwiseconv_kernel(float* input, float* kernel, float* output, int input_height, int kernel_size, int input_channels) {
    float *d_input, *d_kernel, *d_output;
    int input_size = input_height * input_height * input_channels;
    int output_height = input_height - kernel_size + 1;
    int output_width = input_height - kernel_size + 1;
    int filter_size = kernel_size * kernel_size * input_channels;
    int output_size = output_height * output_width * input_channels;
    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_kernel, filter_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, filter_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    depthwise_convolution<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_kernel);
    hipFree(d_output);
}
