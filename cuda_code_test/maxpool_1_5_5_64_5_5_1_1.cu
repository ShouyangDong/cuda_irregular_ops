#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(1024) maxpool(float* __restrict__ A, float* __restrict__ pool_max) {
  float pool_max_local[1];
  pool_max_local[0] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      if (((int)threadIdx.x) < 64) {
        pool_max_local[0] = max(pool_max_local[0], A[(((rv0 * 320) + (rv1 * 64)) + ((int)threadIdx.x))]);
      }
    }
  }
  if (((int)threadIdx.x) < 64) {
    pool_max[((int)threadIdx.x)] = pool_max_local[0];
  }
}

extern "C" void maxpool_kernel(float *output, float *input, int batch_size, int channels, int input_size, int kernel_size, int stride) {
    float *d_input, *d_output;
    int output_H = (H - kernel_size) / stride + 1;
    int input_size = batch_size * kernel_size * kernel_size * channels;
    int output_size = batch_size * output_H * output_H * channels;
    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, output_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    avgpool<<<numBlocks, blockSize>>>(d_input, d_output);

    hipMemcpy(output, d_output, output_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
