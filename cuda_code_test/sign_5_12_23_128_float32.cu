
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) sign_kernel(float* __restrict__ A, float* __restrict__ T_sign) {
  if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 176640) {
    T_sign[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = ((0.000000e+00f < A[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]) ? 1.000000e+00f : ((A[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] < 0.000000e+00f) ? -1.000000e+00f : 0.000000e+00f));
  }
}