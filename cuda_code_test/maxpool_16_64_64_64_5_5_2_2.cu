#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(1024) maxpool(float* __restrict__ A, float* __restrict__ pool_max) {
  float pool_max_local[1];
  pool_max_local[0] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      pool_max_local[0] = max(pool_max_local[0], A[(((((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) / 225) * 262144) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7)) % 450) / 15) * 8192)) + (rv0 * 4096)) + ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) % 30) * 128)) + (rv1 * 64)) + (((int)threadIdx.x) & 63))]);
    }
  }
  pool_max[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pool_max_local[0];
}

extern "C" void maxpool_kernel(float *output, float *input, int input_size, int kernel_size, int stride) {
    int input_size = 128;
    int kernel_size = 3;
    float *d_input, *d_output;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    maxpool<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
