#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(1024) minpool(float* __restrict__ A, float* __restrict__ pool_min) {
  float pool_min_local[1];
  pool_min_local[0] = 3.402823e+38f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      pool_min_local[0] = min(pool_min_local[0], A[(((((((((int)blockIdx.x) / 81) * 802816) + (((((((int)blockIdx.x) % 81) * 4) + (((int)threadIdx.x) >> 8)) / 9) * 21504)) + (rv0 * 7168)) + ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) % 36) * 192)) + (rv1 * 64)) + (((int)threadIdx.x) & 63))]);
    }
  }
  pool_min[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pool_min_local[0];
}

extern "C" void minpool_kernel(float *output, float *input, int input_size, int kernel_size, int stride) {
    int input_size = 128;
    int kernel_size = 3;
    float *d_input, *d_output;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    minpool<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
