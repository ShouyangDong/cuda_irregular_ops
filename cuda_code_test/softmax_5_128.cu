
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(5) softmax(float* __restrict__ A, float* __restrict__ T_softmax_norm) {
    if (threadIdx.x < 5) {
    int rowStart = threadIdx.x * 128;
    
    float maxVal = A[rowStart];
    for (int i = 1; i < 128; ++i) {
        if (A[rowStart + i] > maxVal) {
            maxVal = A[rowStart + i];
        }
    }
    
    float denom = 0.0f;
    for (int i = 0; i < 128; ++i) {
        T_softmax_norm[rowStart + i] = expf(A[rowStart + i] - maxVal);
        denom += T_softmax_norm[rowStart + i];
    }
    
    for (int i = 0; i < 128; ++i) {
        T_softmax_norm[rowStart + i] /= denom;
    }
  }
}


extern "C" void softmax_kernel(float *C, float *A, int size) {
  float *d_A, *d_C;

  hipMalloc(&d_A, size * sizeof(float));
  hipMalloc(&d_C, size * sizeof(float));

  hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(1024);
  dim3 numBlocks((size + 1024 - 1) / 1024);

  softmax<<<numBlocks, blockSize>>>(d_A, d_C);

  hipMemcpy(C, d_C, size * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
