#include "hip/hip_runtime.h"
__global__ void gemv(float *y, float *A, float *x) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < 32) {
        float sum = 0.0f;
        for (int i = 0; i < 512; i++) {
            sum += A[row * 512 + i] * x[i];
        }
        y[row] = sum;
    }
}

extern "C" void gemv_kernel(float *y, float *A, float *x, int m, int n) {
    float *d_A, *d_x, *d_y;

    hipMalloc(&d_A, m * n * sizeof(float));
    hipMalloc(&d_x, n * sizeof(float));
    hipMalloc(&d_y, m * sizeof(float));

    hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (m + blockSize - 1) / blockSize;

    gemv<<<numBlocks, blockSize>>>(d_A, d_x, d_y, m, n);

    hipMemcpy(y, d_y, m * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
}
