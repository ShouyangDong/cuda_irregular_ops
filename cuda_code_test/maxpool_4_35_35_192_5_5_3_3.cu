#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(1024) maxpool(float* __restrict__ A, float* __restrict__ pool_max) {
  float pool_max_local[1];
  pool_max_local[0] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 363) {
        pool_max_local[0] = max(pool_max_local[0], A[(((((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 363) * 235200) + (((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) % 363) / 33) * 20160)) + (rv0 * 6720)) + (((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) % 33) / 3) * 576)) + (rv1 * 192)) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 192))]);
      }
    }
  }
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 363) {
    pool_max[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pool_max_local[0];
  }
}

extern "C" void maxpool_kernel(float *output, float *input, int input_size, int kernel_size, int stride) {
    int input_size = 128;
    int kernel_size = 3;
    float *d_input, *d_output;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    maxpool<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
