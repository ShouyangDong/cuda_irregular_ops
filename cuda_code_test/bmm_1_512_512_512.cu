
#include <hip/hip_runtime.h>
__global__ void bmm(float *A, float *B, float *C) {
    int batch_idx = blockIdx.z;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (batch_idx < 1 && row < 512 && col < 512) {
        float sum = 0.0f;
        for (int i = 0; i < 512; i++) {
            sum += A[batch_idx * 512 * 512 + row * 512 + i] * B[batch_idx * 512 * 512 + i * 512 + col];
        }
        C[batch_idx * 512 * 512 + row * 512 + col] = sum;
    }
}

extern "C" void bmm_kernel(float *C, float *A, float *B, int b, int m, int k, int n) {
    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(512, 512, 1);
    dim3 numBlocks((m + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y, b + blockSize.z - 1);

    bmm<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
