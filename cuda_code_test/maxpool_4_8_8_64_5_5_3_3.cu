#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(1024) maxpool(float* __restrict__ A, float* __restrict__ pool_max) {
  float pool_max_local[1];
  pool_max_local[0] = -3.402823e+38f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      pool_max_local[0] = max(pool_max_local[0], A[(((((((((int)threadIdx.x) >> 8) * 4096) + (((((int)threadIdx.x) & 255) >> 7) * 1536)) + (rv0 * 512)) + (((((int)threadIdx.x) & 127) >> 6) * 192)) + (rv1 * 64)) + (((int)threadIdx.x) & 63))]);
    }
  }
  pool_max[((int)threadIdx.x)] = pool_max_local[0];
}

extern "C" void maxpool_kernel(float *output, float *input, int input_size, int kernel_size, int stride) {
    int input_size = 128;
    int kernel_size = 3;
    float *d_input, *d_output;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    maxpool<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
