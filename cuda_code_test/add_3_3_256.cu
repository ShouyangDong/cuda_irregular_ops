#include "hip/hip_runtime.h"
extern "C" __global__ void __launch_bounds__(1024) add(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add) {
  if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 2309) {
    T_add[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (A[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + B[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}

extern "C" void add_kernel(float *C, float *A, float *B, int size) {
  float *d_A, *d_B, *d_C;

  hipMalloc(&d_A, size * sizeof(float));
  hipMalloc(&d_B, size * sizeof(float));
  hipMalloc(&d_C, size * sizeof(float));

  hipMemcpy(d_A, A, size * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, size * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(128, 128);
  dim3 numBlocks((m + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

  add<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

  hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
