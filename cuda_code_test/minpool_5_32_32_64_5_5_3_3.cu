#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(1024) minpool(float* __restrict__ A, float* __restrict__ pool_min) {
  float pool_min_local[1];
  pool_min_local[0] = 3.402823e+38f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 125) {
        pool_min_local[0] = min(pool_min_local[0], A[(((((((((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) / 25) * 65536) + (((((((int)blockIdx.x) * 8) + (((int)threadIdx.x) >> 7)) % 50) / 5) * 6144)) + (rv0 * 2048)) + ((((((int)blockIdx.x) * 6) + (((int)threadIdx.x) >> 6)) % 10) * 192)) + (rv1 * 64)) + (((int)threadIdx.x) & 63))]);
      }
    }
  }
  if (((((int)blockIdx.x) * 4) + (((int)threadIdx.x) >> 8)) < 125) {
    pool_min[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pool_min_local[0];
  }
}

extern "C" void minpool_kernel(float *output, float *input, int input_size, int kernel_size, int stride) {
    int input_size = 128;
    int kernel_size = 3;
    float *d_input, *d_output;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    minpool<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
