#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(1024) sumpool(float* __restrict__ A, float* __restrict__ pool_avg) {
  float pool_sum[1];
  pool_sum[0] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      if (((int)threadIdx.x) < 256) {
        pool_sum[0] = (pool_sum[0] + A[((((((((int)threadIdx.x) >> 7) * 640) + (rv0 * 320)) + (((((int)threadIdx.x) & 127) >> 6) * 128)) + (rv1 * 64)) + (((int)threadIdx.x) & 63))]);
      }
    }
  }
  if (((int)threadIdx.x) < 256) {
    pool_avg[((int)threadIdx.x)] = pool_sum[0];
  }
}

extern "C" void sumpool_kernel(float *output, float *input, int input_size, int kernel_size, int stride) {
    int input_size = 128;
    int kernel_size = 3;
    float *d_input, *d_output;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    sumpool<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}