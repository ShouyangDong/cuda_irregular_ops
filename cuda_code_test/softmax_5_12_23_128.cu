
#include <hip/hip_runtime.h>

__global__ void __launch_bounds__(1024) softmax_kernel(float* __restrict__ A, float* __restrict__ T_softmax_exp) {
  int idx = blockIdx.x * 1024 + threadIdx.x;
  if (idx < 1380) {

      float maxVal = A[idx * 128];
      for (int i = 1; i < 128; ++i) {
          if (A[threadIdx.x* 128 + i] > maxVal) {
              maxVal = A[threadIdx.x* 128 + i];
          }
      }
      
      
      float denom = 0.0f;
      for (int i = 0; i < 128; ++i) {
          T_softmax_exp[idx * 128 + i] = expf(A[idx * 128 + i] - maxVal);
          denom += T_softmax_exp[idx * 128 + i];
      }
      
      
      for (int i = 0; i < 128; ++i) {
          T_softmax_exp[idx * 128 + i] /= denom;
      }
  }
}
