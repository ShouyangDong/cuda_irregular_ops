
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(1024) sumpool_kernel(float* __restrict__ A, float* __restrict__ pool_avg) {
  float pool_sum[1];
  pool_sum[0] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 3; ++rv0) {
    for (int rv1 = 0; rv1 < 3; ++rv1) {
      if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 15125) {
        pool_sum[0] = (pool_sum[0] + A[(((((((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) / 3025) * 802816) + (((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) % 3025) / 55) * 14336)) + (rv0 * 7168)) + ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) % 55) * 128)) + (rv1 * 64)) + (((int)threadIdx.x) & 63))]);
      }
    }
  }
  if (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) < 15125) {
    pool_avg[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pool_sum[0];
  }
}