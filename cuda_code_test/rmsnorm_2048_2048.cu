
#include <hip/hip_runtime.h>
__global__ void cuda_rms_norm(float* A, float* B) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float eps = 1e-5f;

  if (idx < 2048) {
    // Calculate sum
    float sum = 0.0;
    for (int j = 0; j < 2048; j++) {
      sum += A[idx * 2048 + j] * A[idx * 2048 + j];
    }

    // Calculate mean
    float mean = sum / 2048;

    // Calculate scale
    float scale = 1.0 / sqrt(mean + eps);

    // Normalize and store in B
    for (int j = 0; j < 2048; j++) {
      B[idx * 2048 + j] = A[idx * 2048 + j] * scale;
    }
  }
}

extern "C" void rms_norm_kernel(float* A, float* B) {
  // Allocate memory on the device
  float *d_A, *d_B;
  int size = 2048;
  int num_elements = size * size;
  hipMalloc(&d_A, num_elements * sizeof(float));
  hipMalloc(&d_B, num_elements * sizeof(float));

  // Copy data from host to device
  hipMemcpy(d_A, A, num_elements * sizeof(float), hipMemcpyHostToDevice);

  // Define grid and block dimensions
  int block_size = 256;
  int num_blocks = (size + block_size - 1) / block_size;

  // Launch kernel
  cuda_rms_norm<<<num_blocks, block_size>>>(d_A, d_B);

  // Copy the result back to host
  hipMemcpy(B, d_B, num_elements * sizeof(float), hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(d_A);
  hipFree(d_B);
}
