
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(45) softmax_kernel(float* __restrict__ A, float* __restrict__ T_softmax_norm) {
  if (threadIdx.x < 45) {
    int rowStart = threadIdx.x * 25;
    
    float maxVal = A[rowStart];
    for (int i = 1; i < 25; ++i) {
        if (A[rowStart + i] > maxVal) {
            maxVal = A[rowStart + i];
        }
    }
    
    float denom = 0.0f;
    for (int i = 0; i < 25; ++i) {
        T_softmax_norm[rowStart + i] = expf(A[rowStart + i] - maxVal);
        denom += T_softmax_norm[rowStart + i];
    }
    
    for (int i = 0; i < 25; ++i) {
        T_softmax_norm[rowStart + i] /= denom;
    }
  }
}
