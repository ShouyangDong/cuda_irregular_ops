
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) add_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add) {
  if (((blockIdx.x * 1024) + (threadIdx.x)) < 4096) {
    T_add[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (A[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + B[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}