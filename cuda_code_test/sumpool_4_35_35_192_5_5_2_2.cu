#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(1024) sumpool(float* __restrict__ A, float* __restrict__ pool_avg) {
  float pool_sum[1];
  pool_sum[0] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      pool_sum[0] = (pool_sum[0] + A[(((((((((int)blockIdx.x) / 48) * 235200) + (((((int)blockIdx.x) % 48) / 3) * 13440)) + (rv0 * 6720)) + (((((((int)blockIdx.x) % 3) * 16) + (((int)threadIdx.x) >> 6)) / 3) * 384)) + (rv1 * 192)) + (((((int)blockIdx.x) * 64) + ((int)threadIdx.x)) % 192))]);
    }
  }
  pool_avg[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = pool_sum[0];
}

extern "C" void sumpool_kernel(float *output, float *input, int input_size, int kernel_size, int stride) {
    int input_size = 128;
    int kernel_size = 3;
    float *d_input, *d_output;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    sumpool<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}
