
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) add_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ T_add) {
  if (((((int)blockIdx.x) * 1024) + ((int)threadIdx.x)) < 4032) {
    T_add[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (A[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] + B[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))]);
  }
}