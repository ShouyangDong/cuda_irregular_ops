
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(42) softmax_kernel(float* __restrict__ A, float* __restrict__ T_softmax_norm) {
    if (threadIdx.x < 42) {
    int rowStart = threadIdx.x * 7;
    
    float maxVal = A[rowStart];
    for (int i = 1; i < 7; ++i) {
        if (A[rowStart + i] > maxVal) {
            maxVal = A[rowStart + i];
        }
    }
    
    float denom = 0.0f;
    for (int i = 0; i < 7; ++i) {
        T_softmax_norm[rowStart + i] = expf(A[rowStart + i] - maxVal);
        denom += T_softmax_norm[rowStart + i];
    }
    
    for (int i = 0; i < 7; ++i) {
        T_softmax_norm[rowStart + i] /= denom;
    }
  }
}
