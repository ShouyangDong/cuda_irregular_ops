#include "hip/hip_runtime.h"
__global__ void __launch_bounds__(1024) avgpool(float* __restrict__ A, float* __restrict__ pool_avg) {
  float pool_sum[1];
  pool_sum[0] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      pool_sum[0] = (pool_sum[0] + A[(((((((((int)blockIdx.x) / 81) * 802816) + (((((((int)blockIdx.x) % 81) * 4) + (((int)threadIdx.x) >> 8)) / 9) * 21504)) + (rv0 * 7168)) + ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) % 36) * 192)) + (rv1 * 64)) + (((int)threadIdx.x) & 63))]);
    }
  }
  pool_avg[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pool_sum[0] * 4.000000e-02f);
}

extern "C" void avgpool_kernel(float *output, float *input, int input_size, int kernel_size, int stride) {
    int input_size = 128;
    int kernel_size = 3;
    float *d_input, *d_output;

    hipMalloc(&d_input, input_size * sizeof(float));
    hipMalloc(&d_output, input_size * sizeof(float));

    hipMemcpy(d_input, input, input_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(128);
    dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

    avgpool<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);

    hipMemcpy(output, d_output, input_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}