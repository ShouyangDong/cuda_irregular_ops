
#include <hip/hip_runtime.h>
extern "C" __global__ void __launch_bounds__(1024) avgpool_kernel(float* __restrict__ A, float* __restrict__ pool_avg) {
  float pool_sum[1];
  pool_sum[0] = 0.000000e+00f;
  for (int rv0 = 0; rv0 < 5; ++rv0) {
    for (int rv1 = 0; rv1 < 5; ++rv1) {
      pool_sum[0] = (pool_sum[0] + A[(((((((((int)blockIdx.x) / 81) * 802816) + (((((((int)blockIdx.x) % 81) * 4) + (((int)threadIdx.x) >> 8)) / 9) * 21504)) + (rv0 * 7168)) + ((((((int)blockIdx.x) * 16) + (((int)threadIdx.x) >> 6)) % 36) * 192)) + (rv1 * 64)) + (((int)threadIdx.x) & 63))]);
    }
  }
  pool_avg[((((int)blockIdx.x) * 1024) + ((int)threadIdx.x))] = (pool_sum[0] * 4.000000e-02f);
}