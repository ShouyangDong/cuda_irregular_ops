#include "hip/hip_runtime.h"
__global__ void gemm_kernel(float *A, float *B, float *C) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < 4 && col < 6) {
        float sum = 0.0f;
        for (int i = 0; i < 16; i++) {
            sum += A[row * 16 + i] * B[i * 6 + col];
        }
        C[row * 6 + col] = sum;
    }
}

extern "C" void gemm(float *C, float *A, float *B, int m, int k, int n) {
    int m = 4;
    int n = 6;
    int k = 16;

    float *d_A, *d_B, *d_C;

    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 numBlocks((m + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

    gemm_kernel<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
