#include <hip/hip_runtime.h>
#include <random>
#include <vector>
#include <cmath>
#include <iostream>

const int BLOCK_SIZE = 16;

__global__ void fusedScaledDotProductAttention(
    const float* Q, 
    const float* K,
    const float* V,
    float* attention, 
    int sequenceLength, 
    int dim
    ) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    __shared__ float expValues[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sumOfExponents[BLOCK_SIZE];

    if (threadIdx.x == 0) {
        sumOfExponents[threadIdx.y] = 0.0f;
    }

    __syncthreads();

    if (row < sequenceLength && col < dim) {
        float dotProduct = 0.0f;

        // Compute the dot product Q * K^T
        for (int idx = 0; idx < dim; ++idx) {
            dotProduct += Q[row * dim + idx] * K[idx * sequenceLength + col];
        }

        // Scale the dot product
        dotProduct /= sqrtf(static_cast<float>(dim));

        // Apply exponential function
        float expValue = expf(dotProduct);
        expValues[threadIdx.y][threadIdx.x] = expValue;

        // Accumulate the sum of exponentials
        atomicAdd(&sumOfExponents[threadIdx.y], expValue);
    }

    __syncthreads();

    if (row < sequenceLength && col < dim) {
        // Normalize the exponentiated values
        float softmaxValue = expValues[threadIdx.y][threadIdx.x] / sumOfExponents[threadIdx.y];

        // Compute the final attention value
        float finalValue = 0.0f;
        for (int idx = 0; idx < dim; ++idx) {
            finalValue += softmaxValue * V[idx * sequenceLength + col];
        }

        // Store the final attention value
        attention[row * dim + col] = finalValue;
    }
}

void fillMatrix(float* matrix, int rows, int cols) {
    for (int row = 0; row < rows; ++row) {
        for (int col = 0; col < cols; ++col) {
            matrix[row * cols + col] = static_cast<float>(rand()) / RAND_MAX;
        }
    }
}

int main() {
    int sequenceLength = 512;
    int dim = 1024;
    size_t size = sequenceLength * dim * sizeof(float);

    float *queryProjection, *keyProjection, *valueProjection, *attention;
    hipMalloc(&queryProjection, size);
    hipMalloc(&keyProjection, size);
    hipMalloc(&valueProjection, size);
    hipMalloc(&attention, size);

    // Create host arrays and fill them
    float *hostQueryProjection = (float*)malloc(size);
    float *hostKeyProjection = (float*)malloc(size);
    float *hostValueProjection = (float*)malloc(size);
    
    fillMatrix(hostQueryProjection, sequenceLength, dim);
    fillMatrix(hostKeyProjection, sequenceLength, dim);
    fillMatrix(hostValueProjection, sequenceLength, dim);

    // Copy data from host to device
    hipMemcpy(queryProjection, hostQueryProjection, size, hipMemcpyHostToDevice);
    hipMemcpy(keyProjection, hostKeyProjection, size, hipMemcpyHostToDevice);
    hipMemcpy(valueProjection, hostValueProjection, size, hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((dim + BLOCK_SIZE - 1) / BLOCK_SIZE, (sequenceLength + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch the kernel
    fusedScaledDotProductAttention<<<gridDim, blockDim>>>(
        queryProjection,
        keyProjection,
        valueProjection,
        attention,
        sequenceLength,
        dim
    );

    // Check for errors
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Free device memory
    hipFree(queryProjection);
    hipFree(keyProjection);
    hipFree(valueProjection);
    hipFree(attention);

    // Free host memory
    free(hostQueryProjection);
    free(hostKeyProjection);
    free(hostValueProjection);

    return 0;
}