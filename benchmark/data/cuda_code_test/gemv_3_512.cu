
#include <hip/hip_runtime.h>
__global__ void gemv(float *y, float *A, float *x) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  if (row < 3) {
    float sum = 0.0f;
    for (int i = 0; i < 512; i++) {
      sum += A[row * 512 + i] * x[i];
    }
    y[row] = sum;
  }
}

extern "C" void gemv_kernel(float *y, float *A, float *x, int m, int n) {
  float *d_A, *d_x, *d_y;

  hipMalloc(&d_A, m * n * sizeof(float));
  hipMalloc(&d_x, n * sizeof(float));
  hipMalloc(&d_y, m * sizeof(float));

  hipMemcpy(d_A, A, m * n * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

  int blockSize = 3;
  int numBlocks = (m + blockSize - 1) / blockSize;

  for (int i = 0; i < 1000; ++i) {
    gemv<<<numBlocks, blockSize>>>(d_y, d_A, d_x);
  }
  // 定义 CUDA 事件以计算时间
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // 启动内核
  hipEventRecord(start);
  for (int i = 0; i < 1000; ++i) {
    gemv<<<numBlocks, blockSize>>>(d_y, d_A, d_x);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // 计算执行时间
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  milliseconds = milliseconds / 1000.0f;
  printf("Execution time: %f milliseconds\n", milliseconds);

  hipMemcpy(y, d_y, m * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_x);
  hipFree(d_y);
}
