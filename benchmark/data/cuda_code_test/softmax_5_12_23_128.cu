
#include <hip/hip_runtime.h>

__global__ void __launch_bounds__(1024)
    softmax(float *__restrict__ A, float *__restrict__ T_softmax_exp) {
  int idx = blockIdx.x * 1024 + threadIdx.x;
  if (idx < 1380) {

    float maxVal = A[idx * 128];
    for (int i = 1; i < 128; ++i) {
      if (A[threadIdx.x * 128 + i] > maxVal) {
        maxVal = A[threadIdx.x * 128 + i];
      }
    }

    float denom = 0.0f;
    for (int i = 0; i < 128; ++i) {
      T_softmax_exp[idx * 128 + i] = expf(A[idx * 128 + i] - maxVal);
      denom += T_softmax_exp[idx * 128 + i];
    }

    for (int i = 0; i < 128; ++i) {
      T_softmax_exp[idx * 128 + i] /= denom;
    }
  }
}

extern "C" void softmax_kernel(float *C, float *A, int size1, int size2) {
  float *d_A;
  float *d_C;

  hipMalloc(&d_A, size1 * size2 * sizeof(float));
  hipMalloc(&d_C, size1 * size2 * sizeof(float));

  hipMemcpy(d_A, A, size1 * size2 * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(1024);
  dim3 numBlocks((size1 + 1024 - 1) / 1024);

  for (int i = 0; i < 10; i++) {
    softmax<<<numBlocks, blockSize>>>(d_A, d_C);
  }

  // 定义 CUDA 事件以计算时间
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // 启动内核
  hipEventRecord(start);
  for (int i = 0; i < 1000; ++i) {
    softmax<<<numBlocks, blockSize>>>(d_A, d_C);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // 计算执行时间
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  milliseconds = milliseconds / 1000.0f;
  printf("Execution time: %f milliseconds\n", milliseconds);

  hipMemcpy(C, d_C, size1 * size2 * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
