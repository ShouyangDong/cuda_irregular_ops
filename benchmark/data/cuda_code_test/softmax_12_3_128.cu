
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(36)
    softmax(float *__restrict__ A, float *__restrict__ T_softmax_exp) {
  if (threadIdx.x < 36) {

    float maxVal = A[threadIdx.x * 128];
    for (int i = 1; i < 128; ++i) {
      if (A[threadIdx.x * 128 + i] > maxVal) {
        maxVal = A[threadIdx.x * 128 + i];
      }
    }

    float denom = 0.0f;
    for (int i = 0; i < 128; ++i) {
      T_softmax_exp[threadIdx.x * 128 + i] =
          expf(A[threadIdx.x * 128 + i] - maxVal);
      denom += T_softmax_exp[threadIdx.x * 128 + i];
    }

    for (int i = 0; i < 128; ++i) {
      T_softmax_exp[threadIdx.x * 128 + i] /= denom;
    }
  }
}

extern "C" void softmax_kernel(float *C, float *A, int size1, int size2) {
  float *d_A;
  float *d_C;

  hipMalloc(&d_A, size1 * size2 * sizeof(float));
  hipMalloc(&d_C, size1 * size2 * sizeof(float));

  hipMemcpy(d_A, A, size1 * size2 * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(36);
  dim3 numBlocks((size1 + 36 - 1) / 36);

    for (int i =0; i< 10; i++){
    softmax<<<numBlocks, blockSize>>>(d_A, d_C);
  }
  
  // 定义 CUDA 事件以计算时间
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // 启动内核
  hipEventRecord(start);
  for (int i = 0; i < 1000; ++i) {
      softmax<<<numBlocks, blockSize>>>(d_A, d_C);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // 计算执行时间
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  milliseconds = milliseconds / 1000.0f;
  printf("Execution time: %f milliseconds\n", milliseconds);

  hipMemcpy(C, d_C, size1 * size2 * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
