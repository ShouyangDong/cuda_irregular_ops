
#include <hip/hip_runtime.h>
__global__ void conv1d(float *input, float *kernel, float *output) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < 25) {
    output[idx] = 0;
    for (int j = 0; j < 3; j++) {
      output[idx] += input[idx + j] * kernel[j];
    }
  }
}

extern "C" void conv1d_kernel(float *output, float *input, float *kernel,
                              int input_size, int output_size) {
  float *d_input, *d_kernel, *d_output;
  int kernel_size = input_size - output_size + 1;
  hipMalloc(&d_input, input_size * sizeof(float));
  hipMalloc(&d_kernel, kernel_size * sizeof(float));
  hipMalloc(&d_output, output_size * sizeof(float));

  hipMemcpy(d_input, input, input_size * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_kernel, kernel, kernel_size * sizeof(float),
             hipMemcpyHostToDevice);

  dim3 blockSize(25);
  dim3 numBlocks((input_size + blockSize.x - 1) / blockSize.x);

      for (int i =0; i< 10; i++){
    conv1d<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);
  }
  
  // 定义 CUDA 事件以计算时间
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // 启动内核
  hipEventRecord(start);
  for (int i = 0; i < 1000; ++i) {
      conv1d<<<numBlocks, blockSize>>>(d_input, d_kernel, d_output);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // 计算执行时间
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  milliseconds = milliseconds / 1000.0f;
  printf("Execution time: %f milliseconds\n", milliseconds);

  hipMemcpy(output, d_output, output_size * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_kernel);
  hipFree(d_output);
}
