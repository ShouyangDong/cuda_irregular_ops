
#include <hip/hip_runtime.h>
__global__ void bmm(float *A, float *B, float *C) {
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;
  for (int batch_idx = 0; batch_idx < 4; ++batch_idx) {
    if (row < 256 && col < 256) {
      float sum = 0.0f;
      for (int i = 0; i < 256; i++) {
        sum += A[batch_idx * 256 * 256 + row * 256 + i] *
               B[batch_idx * 256 * 256 + i * 256 + col];
      }
      C[batch_idx * 256 * 256 + row * 256 + col] = sum;
    }
  }
}

extern "C" void bmm_kernel(float *C, float *A, float *B, int b, int m, int k,
                           int n) {
  float* d_A;
  float* d_B;
  float* d_C;

  hipMalloc(&d_A, b * m * k * sizeof(float));
  hipMalloc(&d_B, b * k * n * sizeof(float));
  hipMalloc(&d_C, b * m * n * sizeof(float));

  hipMemcpy(d_A, A, b * m * k * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, b * k * n * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(32, 32);
  dim3 numBlocks((m + blockSize.x - 1) / blockSize.x,
                 (n + blockSize.y - 1) / blockSize.y);

  bmm<<<numBlocks, blockSize>>>(d_A, d_B, d_C);

  hipMemcpy(C, d_C, b * m * n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
}
