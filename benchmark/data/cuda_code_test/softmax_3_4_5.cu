
#include <hip/hip_runtime.h>
__global__ void __launch_bounds__(12)
    softmax(float *__restrict__ A, float *__restrict__ T_softmax_maxelem) {
  if (threadIdx.x < 12) {

    float maxVal = A[threadIdx.x * 5];
    for (int i = 1; i < 5; ++i) {
      if (A[threadIdx.x * 5 + i] > maxVal) {
        maxVal = A[threadIdx.x * 5 + i];
      }
    }

    float denom = 0.0f;
    for (int i = 0; i < 5; ++i) {
      T_softmax_maxelem[threadIdx.x * 5 + i] =
          expf(A[threadIdx.x * 5 + i] - maxVal);
      denom += T_softmax_maxelem[threadIdx.x * 5 + i];
    }

    for (int i = 0; i < 5; ++i) {
      T_softmax_maxelem[threadIdx.x * 5 + i] /= denom;
    }
  }
}

extern "C" void softmax_kernel(float *C, float *A, int size1, int size2) {
  float *d_A;
  float *d_C;

  hipMalloc(&d_A, size1 * size2 * sizeof(float));
  hipMalloc(&d_C, size1 * size2 * sizeof(float));

  hipMemcpy(d_A, A, size1 * size2 * sizeof(float), hipMemcpyHostToDevice);

  dim3 blockSize(12);
  dim3 numBlocks((size1 + 12 - 1) / 12);

    for (int i =0; i< 10; i++){
    softmax<<<numBlocks, blockSize>>>(d_A, d_C);
  }
  
  // 定义 CUDA 事件以计算时间
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // 启动内核
  hipEventRecord(start);
  for (int i = 0; i < 1000; ++i) {
      softmax<<<numBlocks, blockSize>>>(d_A, d_C);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);

  // 计算执行时间
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);
  milliseconds = milliseconds / 1000.0f;
  printf("Execution time: %f milliseconds\n", milliseconds);

  hipMemcpy(C, d_C, size1 * size2 * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_A);
  hipFree(d_C);
}
